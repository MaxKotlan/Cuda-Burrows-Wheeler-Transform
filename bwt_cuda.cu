#include "hip/hip_runtime.h"
#include "bwt_cuda.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

struct KernelParameters{
    unsigned char* input;
    unsigned char* output;
    unsigned int*  indices;
    unsigned int  datasize;
};

__device__ void BWTBitonicSort(KernelParameters parameters){

}

__global__ void Main_Kernel_BWT(KernelParameters parameters){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (idx < parameters.datasize){
        /*Initalize Indices to the integers*/
        parameters.indices[idx] = idx;
        __syncthreads();

        /*Sort Indices Using a Bitonic Sort*/
        BWTBitonicSort(parameters);
        __syncthreads();

        /*Convert Input Parameters to Output Parameters Using Sorted Indices*/
        parameters.output[idx] = parameters.input[(-1 - parameters.indices[idx] + parameters.datasize ) % parameters.datasize];
    }
}

TransformedData BWT_CUDA(const std::vector<unsigned char>& input){
    unsigned char* device_input = nullptr; unsigned char* device_output = nullptr; unsigned int* device_indices = nullptr;
    unsigned int k = input.size();
    std::vector<unsigned char> output(k);

    gpuErrchk(hipMalloc((void **)&device_input,   k*sizeof(unsigned char)));
    gpuErrchk(hipMalloc((void **)&device_output,  k*sizeof(unsigned char)));
    gpuErrchk(hipMalloc((void **)&device_indices, k*sizeof(unsigned int)));

    gpuErrchk(hipMemcpy(device_input,   input.data(), k*sizeof(unsigned char), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(device_output, output.data(), k*sizeof(unsigned char), hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);    

    KernelParameters parameters = { device_input, device_output, device_indices, k };
    
    gpuErrchk(hipEventRecord(start));
    unsigned int threadsperblock = 1024;
    Main_Kernel_BWT<<< k/threadsperblock+1, threadsperblock>>>(parameters);
    gpuErrchk(hipEventRecord(stop));
    gpuErrchk(hipEventSynchronize(stop));

    gpuErrchk(hipMemcpy(output.data(), device_output, k*sizeof(unsigned char), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(device_input)); gpuErrchk(hipFree(device_indices)); gpuErrchk(hipFree(device_output));

    for (auto c: output)
        std::cout << c;
    std::cout << std::endl;

    return {};
}
